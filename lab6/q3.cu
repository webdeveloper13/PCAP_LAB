#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void add(int *a, int *b, int *alpha) {
	int i = blockIdx.x;
	b[i]=*alpha*a[i]+b[i];
}


int main(void)
{
	int MAX = 10;
	int a[MAX], b[MAX], alpha;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*MAX;

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, sizeof(int));

	for (int i = 0; i < MAX; ++i)
	{
		a[i] = i+10;
		b[i] = i*20;
	}

	alpha=2;

	printf("Array A:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", a[i]);
	printf("\nArray B:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", b[i]);


	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, &alpha, sizeof(int), hipMemcpyHostToDevice);
    add<<<MAX,1>>>(d_a, d_b, d_c);
	hipMemcpy(&b, d_b, size, hipMemcpyDeviceToHost);
	printf("\nFinal Result:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", b[i]);	
	
	printf("\n");


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
