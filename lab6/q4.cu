#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void sinw(float *a, float *b) {
	int i = blockIdx.x;
	b[i]=sinf(a[i]);
}

int main(void)
{
	int MAX = 10;
	float a[MAX], b[MAX];
	float *d_a, *d_b;
	int size = sizeof(int)*MAX;

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	
	for (int i = 0; i < MAX; ++i)
	{
		a[i] = (3.14/4)*i;
		
	}

	
	printf("Array A:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%f\t", a[i]);
	


	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
    sinw<<<MAX,1>>>(d_a, d_b);
	hipMemcpy(&b, d_b, size, hipMemcpyDeviceToHost);
	printf("\nFinal result:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%f\t", b[i]);	
	
	printf("\n");


	hipFree(d_a);
	hipFree(d_b);
	
	return 0;
}
