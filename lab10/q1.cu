#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<hip/hip_runtime.h>

__global__ void matmul(int *a,int *b,int *c,int WIDTH){
	int rowa=blockIdx.y*blockDim.y+threadIdx.y;
	int colb=blockIdx.x*blockDim.x+threadIdx.x;
	int sum=0;
	for(int i=0;i<WIDTH;i++)
		sum+=a[rowa*WIDTH+i]*b[i*WIDTH+colb];
	c[rowa*WIDTH+colb]=sum;
}

int main(){
	int WIDTH,BLOCK_WIDTH;
	int *matA,*matB,*matSum;
	int *da,*db,*dc;
	printf("enter width of matrix\n");
	scanf("%d",&WIDTH);
	BLOCK_WIDTH=WIDTH/2;
	printf("Enter elements of matrix A\n");
	matA=(int*)malloc(sizeof(int)*WIDTH*WIDTH);
	for(int i=0;i<WIDTH*WIDTH;i++){
		scanf("%d",&matA[i]);
	}
	printf("Enter elements of matrix B\n");
	matB=(int*)malloc(sizeof(int)*WIDTH*WIDTH);
	for(int i=0;i<WIDTH*WIDTH;i++){
		scanf("%d",&matB[i]);
	}
	matSum=(int*)malloc(sizeof(int)*WIDTH*WIDTH);
	hipMalloc((void**)&da,sizeof(int)*WIDTH*WIDTH);
	hipMalloc((void**)&db,sizeof(int)*WIDTH*WIDTH);
	hipMalloc((void**)&dc,sizeof(int)*WIDTH*WIDTH);
	hipMemcpy(da,matA,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
	hipMemcpy(db,matB,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
	int NumBlocks=WIDTH/BLOCK_WIDTH;
	dim3 grid_conf(NumBlocks,NumBlocks);
	dim3 block_conf(BLOCK_WIDTH,BLOCK_WIDTH);
	matmul<<<grid_conf,block_conf>>>(da,db,dc,WIDTH);
	hipMemcpy(matSum,dc,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
	int n=WIDTH;
	int m=WIDTH;
	printf("Result: \n");
	for(int i=0;i<m;i++){
		for(int j=0;j<n;j++){
			printf("%d ",matSum[i*n+j]);
		}
		printf("\n");
	}
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(matA);
	free(matB);
	free(matSum);
	return 0;
}
