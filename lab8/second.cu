#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void reverseWord(char *a , char *b , int size)
{
    int id = threadIdx.x+1;
    int cur = 0;
    int start = 0;
    int end = size;
    int j = 0;
    for(j = 0;j<size;j++)
    {
        if(a[j] == ' ')
        {
            cur++;
            if(cur == id)
            {
                end = j;
                break;
            }
            else
            {
                start = j;
            }
        }
    }
    int i = 0;
    if(start!=0)
    {
        b[start] = ' ';
        j = start+1;
    }
    else
        j = start;
   	i = end-1;
    for(;j<=i;j++,i--)
    {
        b[j] = a[i];
        b[i] = a[j];
    }
   


}


int main()
{
    int n;
    int size;
    char *a = (char*)malloc(sizeof(char)*(30));
    printf("Enter the string \n");
    scanf("%[^\n]%*c", a); 
    printf("Enter number of words \n");
    scanf("%d",&n);
    char *b = (char*)malloc(sizeof(char)*(30));
    char *d_a , *d_b;
    printf("Input String =  %s \n",a);
    size = strlen(a);
    int size1 = sizeof(char)*(size+1);
    int size2 = sizeof(char)*(size+1);
    hipMalloc((void**)&d_a,size1);
    hipMalloc((void**)&d_b,size2);
    hipMemcpy(d_a,a,sizeof(char)*(size+1),hipMemcpyHostToDevice);
    reverseWord<<<1,n>>>(d_a,d_b,size);
    hipMemcpy(b,d_b,size2,hipMemcpyDeviceToHost);
    printf("Output string =  %s \n",b);
    
    hipFree(d_a);
    hipFree(d_b);


}


