#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void countWord(char *a , char *b , unsigned int* d_count , int size , int wordSize)
{
    int id = threadIdx.x+1;
    int cur = 0;
    int start = 0;
    int end = size;
    int j = 0;
    for(j = 0;j<size;j++)
    {
        if(a[j] == ' ')
        {
            cur++;
            if(cur == id)
            {
                end = j;
                break;
            }
            else
            {
                start = j;
            }
        }
    }


    int i = 0;
    if(start!=0)
    {
        j = start+1;
    }
    else
        j = start;
   	i = end-1;


    int k = 0;
    int flag = 1;

   for(k = 0;k<wordSize;k++,j++)
   {
    if(a[j]!=b[k] || j>i)
        {
            flag = -1;
            break;
        }
   }

   if(flag == 1)
    atomicAdd(d_count,1);



}


int main()
{
    int n;
    unsigned int *count = 0,*d_count,*result = 0;
    count = (unsigned int*)malloc(sizeof(unsigned int));
    result = (unsigned int*)malloc(sizeof(unsigned int));
    char *a = (char*)malloc(sizeof(char)*(30));
    int size = sizeof(int);
    printf("Enter the string \n");
    scanf("%[^\n]%*c", a); 
    printf("Enter number of words \n");
    scanf("%d",&n);
    char *b = (char*)malloc(sizeof(char)*(30));
    printf("Enter the word \n");
    scanf("%s",b);
    char *d_a , *d_b;
    printf("Input String =  %s \n",a);
    int wordSize = strlen(b);
    int size1 = sizeof(char)*30;
    int size2 = sizeof(char)*30;
    hipError_t error;
    error = hipMalloc((void**)&d_a,size1);
    if(error != hipSuccess)
    {
        printf("Error in first malloc\n");
        exit(0);
    }
    error = hipMalloc((void**)&d_b,size2);
    if(error != hipSuccess)
    {
        printf("Error in second malloc\n");
        exit(0);
    }
    error = hipMalloc((void**)&d_count,sizeof(unsigned int));
    if(error != hipSuccess)
    {
        printf("Error in third malloc \n");
        exit(0);
    }
    error = hipMemcpy(d_count,count,sizeof(*count),hipMemcpyHostToDevice);
    if(error != hipSuccess)
    {
        printf("Error in first\n");
        printf("Cuda error 2: %s \n",hipGetErrorString(error));
        exit(0);
    }
    error = hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
    if(error != hipSuccess)
    {
        printf("Error in second\n");
        exit(0);
    }
    error = hipMemcpy(d_b,b,size2,hipMemcpyHostToDevice);
    if(error != hipSuccess)
    {
        printf("Error in third\n");
        exit(0);
    }
    size = strlen(a);
    countWord<<<1,n>>>(d_a,d_b,d_count,size,wordSize);
    hipMemcpy(result,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);
    printf("Total occurences of %s = %d \n",b,*result);
    hipFree(d_a);
    hipFree(d_b);


}


