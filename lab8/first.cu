#include"hip/hip_runtime.h"

#include<stdlib.h>
#include<stdio.h>
#include<string.h>


__global__ void multipleStrings(char* a ,  char* b,int size)
{
	int i = threadIdx.x * size;
	int j = 0;
	for(j=0;j<size;j++)
	{
		b[i+j] = a[j];
	}
}

int main()
{
	int n;
	int size;
	printf("Enter the value of n \n");
	scanf("%d",&n);
	printf("Enter the size of the string \n");
	scanf("%d",&size);
	char *a = (char*)malloc(sizeof(char)*(size+1));
	printf("Enter the string \n");
	scanf("%s",a);
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	char *b = (char*)malloc(sizeof(char)*(n*size+1));
	char *d_a , *d_b;
	int size1 = sizeof(char)*(size+1);
	int size2 = sizeof(char)*(size*n+1);
	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size2);
	hipMemcpy(d_a,a,sizeof(char)*(size+1),hipMemcpyHostToDevice);	
	multipleStrings<<<1,n>>>(d_a,d_b,size);
	hipMemcpy(b,d_b,size2,hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("I am alive \n");
	int l = strlen(b);
	printf("string = %s \n",b);
	printf("Time taken = %f \n",elapsedTime);
	hipFree(d_a);
	hipFree(d_b);


}


