#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void reverseWord(char *a , char *b , int size)
{
    int id = threadIdx.x;
    b[size-id] = a[id];
}




int main()
{
    int size;
    char *a = (char*)malloc(sizeof(char)*(30));
    printf("Enter the string \n");
    scanf("%[^\n]%*c", a); 
    char *b = (char*)malloc(sizeof(char)*(30));
    char *d_a , *d_b;
    printf("Input String =  %s \n",a);
    size = strlen(a);
    int size1 = sizeof(char)*(size+1);
    int size2 = sizeof(char)*(size+1);
    hipMalloc((void**)&d_a,size1);
    hipMalloc((void**)&d_b,size2);
    hipMemcpy(d_a,a,sizeof(char)*(size+1),hipMemcpyHostToDevice);
    reverseWord<<<1,size>>>(d_a,d_b,size-1);
    hipMemcpy(b,d_b,size2,hipMemcpyDeviceToHost);
    printf("Output string =  %s \n",b);
    
    hipFree(d_a);
    hipFree(d_b);


}