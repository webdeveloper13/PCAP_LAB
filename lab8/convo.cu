#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
__global__ void convolution(int *I, int *M, int *O,int * w,int *mw){
	int WIDTH = * w;
	int MASK_WIDTH = *mw;
	int i = threadIdx.x;
	float op = 0;
	int si = i-(MASK_WIDTH/2);
	for(int j =0; j<MASK_WIDTH;j++){
		if(si+j >=0 && si+j < WIDTH){
			op+= I[si+j]*M[j];
		}
	}
	O[i]=op;
}

int main(int argc, char const *argv[]){
	int n,m,*d_w,*d_mw;
	printf("Enter the value n and m\n");
	scanf("%d",&n);
	scanf("%d",&m);
	int input[n],output[n],mask[m],*d_i,*d_m,*d_o;
	printf("Enter elements in 1st input array:\n");
	for(int i = 0;i<n;i++){
		scanf("%d",&input[i]);
	}
	printf("Enter elements in 2nd input array:\n");
	for(int i = 0;i<m;i++){
		scanf("%d",&mask[i]);
	}
	hipMalloc((void **)&d_i,sizeof(int)*n);
	hipMalloc((void **)&d_m,sizeof(int)*m);
	hipMalloc((void **)&d_o,sizeof(int)*n);
	hipMalloc((void **)&d_w,sizeof(int));
	hipMalloc((void **)&d_mw,sizeof(int));
	hipMemcpy(d_i,input,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_m,mask,m*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_w,&n,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_mw,&m,sizeof(int),hipMemcpyHostToDevice);
	convolution<<<1,n>>>(d_i,d_m,d_o,d_w,d_mw);
	hipMemcpy(output,d_o,n*sizeof(int),hipMemcpyDeviceToHost);
	for(int i = 0;i<n;i++){
		printf("%d ",output[i]);
	}
	hipFree(d_i);
	hipFree(d_m);
	hipFree(d_o);
	return 0;
}
