#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void add(int *a, int *b, int *c) {
	int i = blockIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void add2(int *a, int *b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void add3(int *a, int *b, int *c, int MAX) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i<MAX)
		c[i] = a[i] + b[i];
	
}

int main(void)
{
	int MAX = 10;
	int a[MAX], b[MAX], c[MAX];
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*MAX;

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	for (int i = 0; i < MAX; ++i)
	{
		a[i] = i+10;
		b[i] = i*20;
	}

	printf("Array A:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", a[i]);
	printf("\nArray B:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", b[i]);


	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	//No of blocks is MAX, No of threads in each block is 1
	add<<<MAX,1>>>(d_a, d_b, d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("\nOutput 1:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", c[i]);	
	add2<<<1, MAX>>>(d_a, d_b, d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("\nOutput 2:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", c[i]);
	add3<<<ceil(MAX/256), 256>>>(d_a, d_b, d_c, MAX);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("\nOutput 3:\n");
	for (int i = 0; i < MAX; ++i)
		printf("%d\t", c[i]);
	printf("\n");


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
