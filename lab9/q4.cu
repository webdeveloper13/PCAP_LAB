
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void transpose(int *a,int *t)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  int flag=0,comp,j=1;  
  if(blockIdx.x==0 || (blockIdx.x+1)%gridDim.x == 0 || threadIdx.x==0 || (threadIdx.x+1)%blockDim.x==0)
    flag=1;
  if(!flag)
    {
    	t[id]=0;
        while(a[id]!=0){
			comp=a[id]%2;
			if(comp)
				comp=0;
			else
				comp=1;
			t[id]+=(comp*j);
			j*=10;
			a[id]/=2;
		}

    }
 else
 {
     t[id]=a[id];
 }
}

int main(void)
{
	int *t,m,n,i,j;
	int *d_a,*d_t,*d_m;
  m=4;
	n=4;
  int a[]={1,2,3,4,5,5,8,8,9,4,10,12,13,14,15,16};
  int size=sizeof(int)*m*n;
	t=(int*)malloc(m*n*sizeof(int));
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	transpose<<<m,n>>>(d_a,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("result vector is:\n");
	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
		{
			printf("%d\t",t[i*n+j] );
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}