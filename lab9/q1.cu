#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void square(int*a , int *t)
{
	int n = threadIdx.x, m=blockIdx.x, size=blockDim.x, size1=gridDim.x;
	int i= m*size+n;
	t[i]=1;
	//int final=0;
	for(int j=0;j<(m+1);j++)
		t[i]*=a[i];
}

int main(int argc, char const *argv[])
{
	int *a,*t,m,n,i,j;

	int *d_a, *d_t;

	printf("enter the value of m \n");
	scanf("%d",&m);
	printf("enter the value of n\n");
	scanf("%d",&n);

	int size= sizeof(int)*m*n;

	a=(int*)malloc(m*n*sizeof(int));
	t=(int*)malloc(m*n*sizeof(int));

	printf("enter the input matrix\n");

	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	square<<<m,n>>>(d_a,d_t);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);

	printf("the result vector is :\n");

	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
			printf("%d\t",t[i*n+j] );

		printf("\n");
	}

	getchar();
	hipFree(d_a);
	hipFree(d_t);



	return 0;
}