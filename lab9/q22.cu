#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void mat_add(int*a , int *b,int *c,int m)
{
	int i,index;
	int col_id=threadIdx.x;
	for(i=0;i<m;i++)
	{
		index=i*blockDim.x+col_id;
		c[index]=a[index]+b[index];
	}
}

int main(int argc, char const *argv[])
{
	int *a,*b,*c,m,n,i,j;

	int *d_a, *d_b,*d_c;

	printf("enter the value of m \n");
	scanf("%d",&m);
	printf("enter the value of n\n");
	scanf("%d",&n);

	int size= sizeof(int)*m*n;

	a=(int*)malloc(m*n*sizeof(int));
	b=(int*)malloc(m*n*sizeof(int));
	c=(int*)malloc(m*n*sizeof(int));
	printf("enter the input1 matrix\n");

	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);

	printf("enter the input2 matrix\n");

		for(i=0;i<m*n;i++)
			scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	mat_add<<<1,n>>>(d_a,d_b,d_c,m);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

	printf("the result vector is :\n");

	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
			printf("%d\t",c[i*n+j] );

		printf("\n");
	}

	getchar();
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);


	return 0;
}
